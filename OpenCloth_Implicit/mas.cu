#include "hip/hip_runtime.h"

#include <stdio.h>
#include <time.h>

//int main(void)
//{
//	// init data
//	int num = 10;
//	int a[10], b[10], c[10];
//	int* a_gpu, * b_gpu, * c_gpu;
//
//	for (int i = 0; i < num; i++)
//	{
//		a[i] = i;
//		b[i] = i * i;
//	}
//
//
//
//	cudaMalloc((void**)&a_gpu, num * sizeof(int));
//	cudaMalloc((void**)&b_gpu, num * sizeof(int));
//	cudaMalloc((void**)&c_gpu, num * sizeof(int));
//
//	// copy data
//	cudaMemcpy(a_gpu, a, num * sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(b_gpu, b, num * sizeof(int), cudaMemcpyHostToDevice);
//
//	addKernel(a_gpu, b_gpu, c_gpu, num);
//
//	//printf("%d + %d = %d\n", a_gpu[0], b_gpu[0], c_gpu[0]);
//
//	// get data
//	cudaMemcpy(c, c_gpu, num * sizeof(int), cudaMemcpyDeviceToHost);
//
//	// visualization
//	for (int i = 0; i < num; i++)
//	{
//		printf("%d + %d = %d\n", a[i], b[i], c[i]);
//	}
//
//	return 0;
//}
//int main()
//{
//	getThreadNum();
//}